/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void latencyGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}

testResult_t latencyInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, rank*sendcount, type, ncclSum, rep, 1, 0));
    int peer = (rank-1+nranks)%nranks;
    TESTCHECK(InitData(args->expected[i], recvcount, peer*recvcount, type, ncclSum, rep, 1, 0));
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place latency
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void latencyGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t latencyRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t*  comm_ptr, hipStream_t stream) {
  ncclComm_t comm = *comm_ptr;
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  
  if(nRanks != 2) {
    printf("Invalid usage of latency script, only two GPUs can be passed, L is sender, R is receiver.\n");
    return testInternalError;
  }

  int senderRank = 0;
  int receiverRank = 1;

  NCCLCHECK(ncclGroupStart());
  if(rank == senderRank) {
    NCCLCHECK(ncclSend(sendbuff, count, type, receiverRank, comm, stream));
  }
  if(rank == receiverRank) {
    NCCLCHECK(ncclRecv(recvbuff, count, type, senderRank, comm, stream));
  }
  NCCLCHECK(ncclGroupEnd());
  
  return testSuccess;
}

struct testColl latencyTest = {
  "latency",
  latencyGetCollByteCount,
  latencyInitData,
  latencyGetBw,
  latencyRunColl
};

void latencyGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  latencyGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t latencyRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &latencyTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    op_count = 1;
    run_ops = &op;
    run_opnames = &opName;
  } else {
    op_count = test_opnum;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], -1));
    }
  }
  return testSuccess;
}

struct testEngine latencyEngine = {
  latencyGetBuffSize,
  latencyRunTest
};

#pragma weak ncclTestEngine=latencyEngine
